#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES

#include "stdafx.h"
#include "Simulation.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <omp.h>
#include <thread>
#include <stdexcept>
#include <>
#include <stdio.h>

int totalTime = 0;

//----------------------------------CONSTRUCTORS / DECONSTRUCTOR------------------------------------------------------

Simulation::Simulation(void) : tempPlanet(nullptr), tempPlanetMass(200), mouseHeldDown(false) //, averageSpeed("averageSpeed.txt")
{
	//Will be added soon
	//if(!averageSpeed.is_open())
	//{
	//	
	//}
}


Simulation::~Simulation(void)
{
}

//---------------------------------------OPERATOR OVERLOADS-----------------------------------------------------------

//-------------------------------------PUBLIC MEMBER FUNCTIONS--------------------------------------------------------

void Simulation::eventHandler(sf::RenderWindow &window)
{
	// Mouse being held down -- Creates a new planet, but the direction and magnitude of its initial velocity has not yet been
	// input from the user.  On mouse release, the initial velocity is set and the planet is updated with the rest.

	if(sf::Mouse::isButtonPressed(sf::Mouse::Left))		//Since there is no function for button released, we must create our own.
	{
		if(!mouseHeldDown)		//code that is executed when the button is first pressed
		{
			mouseHeldDown = true;
			auto position = sf::Mouse::getPosition(window);
			tempPlanet = std::make_shared<Body>(tempPlanetMass, position.x, position.y);
		}
		else    //code that is executed while the button is being pressed
		{
			if(tempPlanet != nullptr)
			{
				auto cursorPosition = sf::Mouse::getPosition(window);
				sf::Vertex line[] =
				{
					sf::Vertex(sf::Vector2f(tempPlanet->xPosition, tempPlanet->yPosition)),
					sf::Vertex(sf::Vector2f(cursorPosition.x, cursorPosition.y))
				};
				window.draw(line, 2, sf::Lines);
			}
		}
	}
	else
	{
		if(mouseHeldDown)		//code that is exected when the button is released
		{
			auto cursorPosition = sf::Mouse::getPosition(window);
			double distance = calculateDistance(tempPlanet->xPosition, tempPlanet->yPosition, cursorPosition.x, cursorPosition.y);
			
			double deltaX = tempPlanet->xPosition - cursorPosition.x;
			double deltaY = tempPlanet->yPosition - cursorPosition.y;

			tempPlanet->setVelocity(deltaX*500/(tempPlanet->mass), deltaY*300/(tempPlanet->mass));

			planetList.push_back(*tempPlanet);
			tempPlanet = nullptr;
			mouseHeldDown = false;
		}
	}

	//increases the mass of the new planet if the up arrow is pressed
	if(sf::Keyboard::isKeyPressed(sf::Keyboard::Up))
	{
		tempPlanetMass += 25;
	}

	//decreases the mass of the new planet if the down arrow is pressed
	if(sf::Keyboard::isKeyPressed(sf::Keyboard::Down))
	{
		if(tempPlanetMass > 25)
			tempPlanetMass -= 25;
	}

}

int Simulation::runOneIteration(sf::RenderWindow &window, sf::Time &elapsed, double gravConst)
{
	eventHandler(window);
	collisionResolution();		//Each of these functions modifies the members of planetList in a specific way.
	
	sumForces(gravConst);
	update(elapsed);

	drawToWindow(window);

	return totalTime;
}

void Simulation::populate(int number, int xMin, int yMin, int xMax, int yMax)		//populates the planetList with randomly generated planets
{

	auto middleX = xMax / 1.5;
	auto middleY = yMax / 1.5;

	//place black hole in centre of planets
	planetList.emplace_back(150, middleX, middleY);

	for (int i = 0; i < number / 5; ++i)
	{
		//place planets around in a circle with varying masses
		planetList.emplace_back((i + 1) * 0.03, middleX + 400 * cos(i), middleY + 400 * sin(i));

		//place planets around in a circle with varying masses
		planetList.emplace_back((i + 1) * 0.03, middleX + 300 * cos(i), middleY + 300 * sin(i));

		//place planets in an inner circle with varying masses
		planetList.emplace_back((i + 1) * 0.03, middleX + 200 * cos(i), middleY + 200 * sin(i));

		//place planets in an inner circle with varying masses
		planetList.emplace_back((i + 1) * 0.03, middleX + 100 * cos(i), middleY + 100 * sin(i));

		//place planets in an inner circle with varying masses
		planetList.emplace_back((i + 1) * 0.03, middleX + 50 * cos(i), middleY + 50 * sin(i));
	}
}

void Simulation::addPlanet(double mass, double xPosition, double yPosition, double xVelocity, double yVelocity)
{
	planetList.emplace_back(mass, xPosition, yPosition);
}

//------------------------------------PRIVATE MEMBER FUNCTIONS--------------------------------------------------------

void Simulation::collisionResolution()
{
	bool collisionDetected = false;
	auto first = planetList.begin();
	while (first != planetList.end())
	{
		for (auto second = planetList.begin(); second != planetList.end(); second++)
		{
			if ((*first) != (*second) && first->hasCollided(*second))
			{
				planetList.push_back((*first) + (*second));
				planetList.erase(second);
				first = planetList.erase(first);			//erase the two collided planets.
				collisionDetected = true;
				break;
			}
		}
		if (collisionDetected)		//If there is a collision, then "first" automatically points to the next value, so there is no need to increment it.
			collisionDetected = false;
		else
			first++;
	}
}

__global__ void couldaddForces(Body *bodies, double *gravConst)
{
	//get block idx
	unsigned int block_idx = blockIdx.x;
	//get thread idx
	unsigned int thread_idx = threadIdx.x;
	//get the number of threeads per block
	unsigned int block_dim = blockDim.x;
	//get the thread's unique ID - (block_idx * block_dim) + thread_idx
	unsigned int idx = (block_idx * block_dim) + thread_idx;
	//calculate forces
	for (int i = 0; i < 20; ++i)
	{
		
		if (idx != i)
		{
			
			auto x_diff = bodies[idx].xPosition- bodies[i].xPosition;
			auto y_diff = bodies[idx].yPosition - bodies[i].yPosition;
			auto distance = sqrtf(x_diff * x_diff + y_diff * y_diff);
			double magnitude = *gravConst * ((bodies[idx].mass)* (bodies[i].mass) / (pow(distance, 2)));

			double deltaX = bodies[i].xPosition - bodies[idx].xPosition;
			double deltaY = bodies[i].yPosition - bodies[idx].yPosition;

			double xAccel = bodies[idx].xAccel;
			double yAccel = bodies[idx].yAccel;

			bodies[idx].xAccel += (magnitude * (deltaX / distance)) / bodies[idx].mass;
			bodies[idx].yAccel += (magnitude * (deltaY / distance)) / bodies[idx].mass;
		}
	}
}

//Adds up all forces from all other bodies.
void Simulation::sumForces(double gravConst)
{
	sf::Clock clock;
	clock.restart();
	//create host memory
	auto data_size = sizeof(Body) * planetList.size();
	auto data_size2 = sizeof(double) * gravConst;
	
	Body *body_buffer;
	double *buffer_double;

	hipMalloc((void**)&body_buffer, data_size);
	hipMalloc((void**)&buffer_double, data_size2);

	hipMemcpy(body_buffer, &planetList[0], data_size, hipMemcpyHostToDevice);
	hipMemcpy(buffer_double, &gravConst, data_size2, hipMemcpyHostToDevice);

	couldaddForces <<<4, 5>>>(body_buffer, buffer_double);

	//sychronise
	hipDeviceSynchronize();

	//read output buffer back to the host
	hipMemcpy(&planetList[0], body_buffer, data_size, hipMemcpyDeviceToHost);

	//clean up resources
	hipFree(body_buffer);
	hipFree(buffer_double);

	totalTime += clock.restart().asMilliseconds();
}

//Updates position and velocity from the new acceleration value, and sets up the acceleration value for the next iteration
void Simulation::update(sf::Time &elapsed)
{
	for(Body &c : planetList)
	{
		c.updateVelocity(elapsed);
		c.updatePosition(elapsed);
		c.zeroAccel();
	}
}

void Simulation::drawToWindow(sf::RenderWindow &window)
{
	for (Body &c : planetList)
	{
		window.draw(c.circle);
	}

	if(tempPlanet != nullptr)
	{
		window.draw(tempPlanet->circle);
	}
}

double Simulation::calculateDistance(double x1, double y1, double x2, double y2)
{
	return sqrt(pow( (x2 - x1), 2) + pow( (y2 - y1), 2));
}